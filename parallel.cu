#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>
#include <math.h>
#include "body.cpp"
/*
CPU
    readInitStateFile
    collisionTest
    
GPU
    calcForce
    calcAccel        1 thread per cell, reduce      OR      1 thread per row 
    integrateStep    1 thread per row
    checkIntersection  1 thread per cell, access global mem to notify if there was collision

*/


const float GRAVITY = 0.000000000066742;

void readInitStateFile(std::string filename,
                       std::vector<std::string> &name,
                       std::vector<std::string> &color,
                       thrust::host_vector<float> &mass,
                       thrust::host_vector<float> &rad,
                       thrust::host_vector<float> &pos_x,
                       thrust::host_vector<float> &pos_y,
                       thrust::host_vector<float> &vel_x,
                       thrust::host_vector<float> &vel_y) {
    std::string elem;

    std::ifstream file(filename);

    while (std::getline(file, elem, ',')) {
        name.push_back(elem);

        std::getline(file, elem, ',');
        color.push_back(elem);

        std::getline(file, elem, ',');
        mass.push_back(std::stof(elem));

        std::getline(file, elem, ',');
        rad.push_back(std::stof(elem));

        std::getline(file, elem, ',');
        pos_x.push_back(std::stof(elem));

        std::getline(file, elem, ',');
        pos_y.push_back(std::stof(elem));

        std::getline(file, elem, ',');
        vel_x.push_back(std::stof(elem));

        std::getline(file, elem);
        vel_y.push_back(std::stof(elem));
    }
}

__device__ float distance(float x1, float y1, float x2, float y2)
{
    return sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));
}

// F = G * m1 * m2 * r / (||r|| ^3)
// Returns each force component, x and y
__device__ float2 calcForce(float a_mass,
                            float a_pos_x,
                            float a_pos_y,
                            float b_mass,
                            float b_pos_x,
                            float b_pos_y) {
    float gForce = (GRAVITY * a_mass * b_mass) / (pow(distance(a_pos_x, a_pos_y, b_pos_x, b_pos_y), 3));
    float2 f = {gForce * (b_pos_x - a_pos_x), gForce * (b_pos_y - a_pos_y)};
    return f;
}

__global__ void calcAccelerations(float* accel_x, 
                                  float* accel_y, 
                                  float* mass,
                                  float* pos_x,
                                  float* pos_y,
                                  float* vel_x,
                                  float* vel_y,
                                  int sz) {
    int r, c;
    r = threadIdx.x + blockIdx.x * blockDim.x;
    while (r < sz) {
        c = threadIdx.y + blockIdx.y * blockDim.y;
        while (c < sz) {
            if (r == c) {
                c += blockDim.y * gridDim.y;
                continue;
            }
            
            int offset = c + r * sz;
            
            float2 f = calcForce(mass[r], pos_x[r], pos_y[r], mass[c], pos_x[c], pos_y[c]);
            
            // Store the acceleration of body a in [r][c]
            accel_x[offset] = f.x / mass[r];
            accel_y[offset] = f.y / mass[r];
            
            c += blockDim.y * gridDim.y;
        }
        r += blockDim.x * gridDim.x;
    }
}

__global__ void integrateStep(float* accel_x, 
                              float* accel_y,
                              float* pos_x,
                              float* pos_y,
                              float* vel_x,
                              float* vel_y,
                              int sz,
                              int deltaTime) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
	while (r < sz) {
	    float ax = 0;
	    float ay = 0;
	    for (int c = 0; c < sz; c++) {
            ax += accel_x[c + r * sz];
            ay += accel_y[c + r * sz];
        }
        // Update velocity
        vel_x[r] += ax * deltaTime;
        vel_y[r] += ay * deltaTime;
	    
	    // Update position
        pos_x[r] += vel_x[r] * deltaTime;
        pos_y[r] += vel_y[r] * deltaTime;
	    
	    r += blockDim.x * gridDim.x;
	}
}

__device__ int checkIntersection(float x1, float y1, float r1, float x2, float y2, float r2)
{
    float distSq = (x1 - x2) * (x1 - x2) +
                   (y1 - y2) * (y1 - y2);
    float radSumSq = (r1 + r2) * (r1 + r2);
    if (distSq == radSumSq) {
        return 1; // Circles touch each other        
    } else if (distSq > radSumSq) {
        return -1; // Circles do not touch each other
    } else {
        return 0; // Circles intersect each other
    }
}

__global__ void checkCollisions(bool* hasCollided,
                                float* rad,
                                float* pos_x,
                                float* pos_y,
                                int sz) {
    int r, c, offset;
    r = threadIdx.x + blockIdx.x * blockDim.x;
    while (r < sz) {
        c = threadIdx.y + blockIdx.y * blockDim.y;
        while (c < sz) {
            if (r == c) {
                c += blockDim.y * gridDim.y;
                continue;
            }
            if (checkIntersection(pos_x[r], pos_y[r], rad[r], pos_x[c], pos_y[c], rad[c]) != -1) {
                offset = c + r * sz;
                hasCollided[offset] = true;
            }
            
            c += blockDim.y * gridDim.y;
        }
        r += blockDim.x * gridDim.x;
    }
}


bool collisionTest(std::vector<std::string> &name,
                   thrust::host_vector<float> &mass,
                   thrust::host_vector<float> &rad,
                   thrust::host_vector<float> &pos_x,
                   thrust::host_vector<float> &pos_y,
                   thrust::host_vector<float> &vel_x,
                   thrust::host_vector<float> &vel_y,
                   float duration) 
{
    // Transfer to GPU
    thrust::device_vector<float> d_mass = mass;
    thrust::device_vector<float> d_rad = rad;
    thrust::device_vector<float> d_pos_x = pos_x;
    thrust::device_vector<float> d_pos_y = pos_y;
    thrust::device_vector<float> d_vel_x = vel_x;
    thrust::device_vector<float> d_vel_y = vel_y;
    
    float* d_mass_ptr = thrust::raw_pointer_cast(d_mass.data());
    float* d_rad_ptr = thrust::raw_pointer_cast(d_rad.data());
    float* d_pos_x_ptr = thrust::raw_pointer_cast(d_pos_x.data());
    float* d_pos_y_ptr = thrust::raw_pointer_cast(d_pos_y.data());
    float* d_vel_x_ptr = thrust::raw_pointer_cast(d_vel_x.data());
    float* d_vel_y_ptr = thrust::raw_pointer_cast(d_vel_y.data());
    
    bool collisionDetected = false;
    int timestepCounter = 0;
    float deltaTime = 1;//0.01 * 24 * 60 * 60; // 1% of a day in seconds

    thrust::host_vector<float> accel_x(mass.size() * mass.size());
    thrust::host_vector<float> accel_y(mass.size() * mass.size());
    thrust::device_vector<float> d_accel_x = accel_x;
    thrust::device_vector<float> d_accel_y = accel_y;
    float* d_accel_x_ptr = thrust::raw_pointer_cast(d_accel_x.data());
    float* d_accel_y_ptr = thrust::raw_pointer_cast(d_accel_y.data());

    // Initial state viz
    // visualize(bodies);

    dim3 threads(10,10);

    while (!collisionDetected && (timestepCounter < duration))
    {
        calcAccelerations<<<1, threads>>>(d_accel_x_ptr, d_accel_y_ptr, d_mass_ptr, d_pos_x_ptr, d_pos_y_ptr, d_vel_x_ptr, d_vel_y_ptr, mass.size());
        thrust::copy(d_accel_x.begin(), d_accel_x.end(), accel_x.begin());
        thrust::copy(d_accel_y.begin(), d_accel_y.end(), accel_y.begin());
        /*
        for (int r = 0; r < mass.size(); r++) {
            for (int c = 0; c < mass.size(); c++) {
                std::cout << "(" << accel_x[c + r * mass.size()] << "," << accel_y[c + r * mass.size()] << ") ";
            }
            std::cout << std::endl;
        }
        */
        
        integrateStep<<<1, 100>>>(d_accel_x_ptr, d_accel_y_ptr, d_pos_x_ptr, d_pos_y_ptr, d_vel_x_ptr, d_vel_y_ptr, mass.size(), deltaTime);
        thrust::copy(d_pos_x.begin(), d_pos_x.end(), pos_x.begin());
        thrust::copy(d_pos_y.begin(), d_pos_y.end(), pos_y.begin());
        thrust::copy(d_vel_x.begin(), d_vel_x.end(), vel_x.begin());
        thrust::copy(d_vel_y.begin(), d_vel_y.end(), vel_y.begin());
        for (int i = 0; i < mass.size(); i++) {
            std::cout << "Body " << name[i] << ": pos(" << pos_x[i] << "," << pos_y[i] << ") vel(" << vel_x[i] << "," << vel_y[i] << ")" << std::endl;
        }
        std::cout << std::endl;
        
        // Visualize
        // visualize(bodies); // iterate through positions of bodies and display them on a coordinate plane
        
        // Check to see if any bodies have the same position
        
        thrust::host_vector<bool> hasCollided(mass.size(), false);
        thrust::device_vector<bool> d_hasCollided = hasCollided;
        bool* d_hasCollided_ptr = thrust::raw_pointer_cast(d_hasCollided.data());
        
        checkCollisions<<<1,threads>>>(d_hasCollided_ptr, d_rad_ptr, d_pos_x_ptr, d_pos_y_ptr, mass.size());
        int numColl = thrust::reduce(d_hasCollided.begin(), d_hasCollided.end(), 0);
        
        // Make sure the calculations are finished on the gpu before moving on
        hipDeviceSynchronize();
        for (int j = 0; j < mass.size(); j++) {
            if(hasCollided[j] == true) {
                std::cout << name[j] << std::endl;
            }
        }
        if (numColl > 0) {
            collisionDetected = true;
        } else {
            // Add time to the timestep counter
            timestepCounter += deltaTime;
        }
    }
    if (collisionDetected) {
        std::cout << "Collision occurred after " << timestepCounter / (60.0 * 60 * 24) << " days" << std::endl;
    }
    return collisionDetected;
}

int main(int argc, char **argv) {
    if (argc < 2) {
        std::cout << "Missing required filename argument" << std::endl;
        return 1;
    }

    std::string filename = argv[1];
    
    std::vector<std::string> name;
    std::vector<std::string> color;
    thrust::host_vector<float> mass;
    thrust::host_vector<float> rad;
    thrust::host_vector<float> pos_x;
    thrust::host_vector<float> pos_y;
    thrust::host_vector<float> vel_x;
    thrust::host_vector<float> vel_y;

    // Load initial state of bodies into separate vectors for each type of data
    readInitStateFile(filename, name, color, mass, rad, pos_x, pos_y, vel_x, vel_y);
    
    // Take in time duration from the user
    float duration;
    std::cout << "Enter the number of years you would like to test: ";
    std::cin >> duration;
    duration = duration * 365 * 24 * 60 * 60; // change duration to seconds

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    bool collision = collisionTest(name, mass, rad, pos_x, pos_y, vel_x, vel_y, duration);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken by function: " << milliseconds << " milliseconds" << std::endl;
    
    if(collision == 1) {
        std::cout << "There was a collision" << std::endl;
    } else {
        std::cout << "There was no collision" << std::endl;
    }

    return 0;
}
